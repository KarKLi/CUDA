#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void HelloWorldFromGPU(void)
{
    printf("Hello world from GPU!\n");
}

int main()
{
    printf("Hello world from CPU!\n");
    HelloWorldFromGPU<<<1,10>>>();
    hipDeviceReset();
    return 0;
}
